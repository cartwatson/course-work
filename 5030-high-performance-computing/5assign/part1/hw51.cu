/**
 * @file hw51.cpp
 * @author Carter Watson
 * @date nov-20-2023
 * @brief CUDA program to convert a color image to gray scale
 * @todo
 * PART 1: CUDA program to convert a color image to gray scale
 * - [X] read the image file
 * - [X] write a kernal function to perform the gray scale conversion using CUDA
 *   - remember to treat the matrices as 1D arrays and use the corresponding linearization of indexes for every memory access you perform
 *   - provide only the program with your comments as your solution. No need to attach input or output files
 * - [X] save the converted image in a binary file named gc.raw
 * 
 * PART 2: GPU Information
 * - Pick a particular GPU (e.g., from the ones available on a CHPC cluster) and report the information
 *   - [X] Max num of threads per SM
 *     - Assume max num of blocks per SM is 8
 * - [ ] choose three different block sizes and explain analytically how the different block sizes should affect the performance of the application made in Part 1
 * - [X] report experimental results using the three different block sizes
 *   - (i.e., run the kernel 200 times to assess the time accurately)
 * 
 * @note This program is only meant to be run on University of Utah CHPC machines
 *   Helper info to compile/run on CHPC
 *     0. ssh to a CHPC machine
 *     1. salloc -n 1 -N 1 -t 0:15:00 -p notchpeak-shared-short -A notchpeak-shared-short --gres=gpu
 *     2. module load nvhpc
 *     3. nvcc -o hw51 hw51.cu
 *     4. srun ./hw51
 */
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>


/**
 * @brief CUDA function to convert a color image to gray scale
 * 
 * @param grayImage (unsigned char *) processed image
 * @param rgbImage (unsigned char *) unprocessed image
 * @param width (int) width of image
 * @param height (int) height of image
 * @param CHANNELS (int) number of channels per pixel
 * 
 * @note This function is directly from lecture notes
 */
__global__
void RGBToGrayscale(unsigned char * grayImage, unsigned char * rgbImage, int width, int height, int CHANNELS)
{
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;

    if (Col < width && Row < height) {
        // get 1D coordinate for the grayscale image
        int grayOffset = Row*width + Col;

        // one can think of the RGB image having
        // CHANNEL times columns of the gray scale image
        int rgbOffset = grayOffset*CHANNELS;
        unsigned char r = rgbImage[rgbOffset ]; // red value for pixel
        unsigned char g = rgbImage[rgbOffset + 1]; // green value for pixel
        unsigned char b = rgbImage[rgbOffset + 2]; // blue value for pixel
        
        // perform the rescaling and store it
        // We multiply by floating point constants
        grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}


/**
 * @brief 
 * 
 * @param h_rgbImage (std::vector<unsigned char>) unprocessed rgb image
 * @param h_grayImage (std::vector<unsigned char>) allocated gray image, passed by reference 
 * @param blocksizeX (int)
 * @param blocksizeY (int)
 * @param NUM_PIXELS (int)
 * @param WIDTH (int)
 * @param HEIGHT (int)
 * @param CHANNELS (int) num of rgb channels
 *
*/
void cudaBlock(std::vector<unsigned char>& h_rgbImage, std::vector<unsigned char>& h_grayImage, int blocksizeX, int blocksizeY, int NUM_PIXELS, int WIDTH, int HEIGHT, int CHANNELS) {
    // cuda init/mem allocation/mem sharing
    unsigned char *d_rgbImage, *d_grayImage;
    hipMalloc((void **)&d_rgbImage, NUM_PIXELS * CHANNELS * sizeof(unsigned char));
    hipMalloc((void **)&d_grayImage, NUM_PIXELS * sizeof(unsigned char));
    hipMemcpy(d_rgbImage, h_rgbImage.data(), NUM_PIXELS * CHANNELS * sizeof(unsigned char), hipMemcpyHostToDevice);

    // define block and grid sizes
    dim3 blockSize(blocksizeX, blocksizeY, 1);
    dim3 gridSize(ceil(WIDTH / blockSize.x), ceil(HEIGHT / blockSize.y), 1);

    // Convert the image to grayscale on device
    RGBToGrayscale<<<gridSize, blockSize>>>(d_grayImage, d_rgbImage, WIDTH, HEIGHT, CHANNELS);

    // sync up cuda and data
    hipDeviceSynchronize();
    hipMemcpy(h_grayImage.data(), d_grayImage, NUM_PIXELS * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // clean up
    hipFree(d_rgbImage);
    hipFree(d_grayImage);
}


/**
* @brief 
* 
* @note the following values are hardcoded
*   - input file name = gc_conv_1024x1024.raw
*   - output file name = gc.raw
*   - image width = 1024
*   - image height = 1024
*   - number of channels = 3
* 
* @return error_code (int) 
* @retval 0 Success
* @retval 1 Invalid number of arguments
*/
int main(int argc, char *argv[]) {
    // sometimes hardcoding variables is okay
    std::string INPUT_FILE = "gc_conv_1024x1024.raw";
    std::string OUTPUT_FILE = "gc.raw";
    const int WIDTH = 1024;
    const int HEIGHT = 1024;
    const int CHANNELS = 3;
    const int NUM_PIXELS = WIDTH * HEIGHT;

    // Check to see if valid # of params provided
    if (argc != 1) {
        std::cout << "ERROR: Inappropriate number of arguments provided!" << std::endl;
        return 1;
    }
    
    // PART 1
    // init image variables
    std::vector<unsigned char> h_rgbImage(NUM_PIXELS * CHANNELS);
    std::vector<unsigned char> h_grayImage(NUM_PIXELS);

    // get image data
    FILE *fp;
    fp = fopen(INPUT_FILE.c_str(), "rb");
    // make sure file exists
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << INPUT_FILE << std::endl;
        return 1;
    }
    // Read in the image file
    fread(&h_rgbImage[0], sizeof(unsigned char), NUM_PIXELS * CHANNELS, fp);
    fclose(fp);

    // run cuda kernal
    cudaBlock(h_rgbImage, h_grayImage, 16, 16, NUM_PIXELS, WIDTH, HEIGHT, CHANNELS);

    // Save the converted image in a binary file named gc.raw
    fp = fopen(OUTPUT_FILE.c_str(), "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << OUTPUT_FILE << std::endl;
        return 1;
    }
    fwrite(&h_grayImage[0], sizeof(unsigned char), NUM_PIXELS, fp);
    fclose(fp);


    // PART 2
    // use device query.cu to get info
        // nvcc -o device_query device_query.cu
        // srun ./device_query
    // Tesla T4
        // Max num of threads per SM = 1024

    // Choose three different block sizes and explain analytically how the different block sizes should affect the performance
    /**
     * 8x8
     *   - Pros: 
     *   - Cons:
     * 16x16
     *   - Pros: 
     *   - Cons:
     * 32x32
     *   - Pros: 
     *   - Cons:
    */
    // Report experimental results using the three different block sizes
    std::vector<int> blockSizes = {8, 16, 32};
    float totalTime, elapsedTime;
    hipEvent_t start, stop;

    // create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < blockSizes.size(); i++) {
        totalTime = 0.0f;
        for (int j = 0; j < 200; j++) {
            hipEventRecord(start, 0);

            // run CUDA kernel
            cudaBlock(h_rgbImage, h_grayImage, blockSizes[i], blockSizes[i], NUM_PIXELS, WIDTH, HEIGHT, CHANNELS);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);

            totalTime += elapsedTime;
        }
        std::cout << "Average time for block size " << blockSizes[i] << ": " << totalTime / 200.0f << " ms" << std::endl;
    }

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
