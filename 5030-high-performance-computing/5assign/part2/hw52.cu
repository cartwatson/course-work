/**
 * @file hw52.cpp
 * @author Carter Watson
 * @date 2023-12-01
 * @brief transpose an image using a variety of methods
 * @todo
 * [X] read the image file
 * [X] write a CPU based serial transpose
 * [ ] write a kernel function to perform a matrix transposition of the input matrix using the GPU global memory
 * [ ] write a kernel function to perform a matrix transposition of the input matrix making use of tiling (using the GPU shared memory)
 * [X] write a function to validate the result of your transposition
 *       - (i.e., on the host CPU, comparing to a CPU based serial transpose) 
 * [X] write code to compute the bandwidth performance of both kernels (1 and 2), report and comment the result you obtained
 * [X] save the transposed matrix to a binary file
 * 
 * @note This program is only meant to be run on University of Utah CHPC machines
 *   Helper info to compile/run on CHPC
 *     0. ssh u1234567@notchpeak.chpc.utah.edu
 *     1. salloc -n 1 -N 1 -t 0:15:00 -p notchpeak-shared-short -A notchpeak-shared-short --gres=gpu
 *     2. module load nvhpc
 *     3. nvcc -o hw52 hw52.cu
 *     4. srun ./hw52
 * 
 * @note Notable Sources
 *   - https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
 */
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>


/**
 * @brief validate image processing
 * 
 * checks two vectors of image data pixel by pixel to verify they are identical
 * 
 * @param validImage (std::vector<unsigned char>&) - vector of valid image data
 * @param checkImage (std::vector<unsigned char>&) - vector of image data to check
 * @param size (int) - size of image
 * 
 * @return bool - true if image processing is valid, otherwise false
 */
bool validateImageProcessing(std::vector<unsigned char>& validImage, std::vector<unsigned char>& checkImage, int size) {
    bool valid = true;
    for (int i = 0; i < size; i++) {
        if (validImage[i] != checkImage[i]) {
            valid = false;
        }
    }
    return valid;
}

/**
 * @brief convert regular image to rotated image using the cpu 
 * 
 * @param inputImage (std::vector<unsigned char>&) - vector of unmodified image data
 * @param outputImage (std::vector<unsigned char>&) - vector of serially modified image data
 * @param numPixels (int) - number of pixels in image
 * @param width (int) - width of image
 * @param height (int) - height of image
 * @param channels (int) - number of channels in image
 * 
 * @return void
 */
void serialTranposeImage(std::vector<unsigned char>& inputImage, std::vector<unsigned char>& outputImage, int numPixels, int width, int height, int channels) {
    // this is whats happening at a basic level
    // for x = 0 to N:
    //    for y = 0 to M:
    //      a_t[x][y] = a[y][x]
    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) { 
            for (int c = 0; c < channels; c++) {
                outputImage[x * height * channels + y * channels + c] = inputImage[y * width * channels + x * channels + c];
            }
        }
    }
}

/**
 * @brief convert regular image to rotated image using the gpu global memory
 * 
 * @param inputImage (unsigned char *) - pointer to unmodified image data
 * @param outputImage (unsigned char *) - pointer to modified image data
 * @param width (int) - width of image
 * @param height (int) - height of image
 * 
 * @return void
*/
__global__
void globalTransposeImage(unsigned char * inputImage, unsigned char * outputImage, int width, int height) {
    // Calculate the global thread index
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the image dimensions
    if (row < height && col < width) {
        // Calculate the linear indices for accessing input and output images
        int inputIndex = row * width + col;
        int outputIndex = col * height + row;

        // Perform the pixel transposition by copying from input to output
        outputImage[outputIndex] = inputImage[inputIndex];
    }
}

/**
 * @brief convert regular image to rotated image using the gpu global memory
 * 
 * @param inputImage (std::vector<unsigned char>) unprocessed image
 * @param outputImage (std::vector<unsigned char>) allocated vector for converted image
 * @param blocksizeX (int)
 * @param blocksizeY (int)
 * @param size (int)
 * @param width (int)
 * @param height (int)
 *
 * @return void
*/
void globalCudaBlock(std::vector<unsigned char>& inputImage, std::vector<unsigned char>& outputImage, int blocksizeX, int blocksizeY, int width, int height, int size) {
    // cuda init/mem allocation/mem sharing
    unsigned char *d_inputImage, *d_outputImage;
    hipMalloc((void **)&d_inputImage, size);
    hipMalloc((void **)&d_outputImage, size);
    hipMemcpy(d_inputImage, inputImage.data(), size, hipMemcpyHostToDevice);

    // define block and grid sizes
    dim3 blockSize(blocksizeX, blocksizeY, 1);
    dim3 gridSize(ceil(width / blockSize.x), ceil(height / blockSize.y), 1);

    // start timer for bandwidth
    std::chrono::_V2::system_clock::time_point start = std::chrono::high_resolution_clock::now();

    // Convert the image to grayscale on device
    globalTransposeImage<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, width, height);

    // sync up cuda and data
    hipDeviceSynchronize();
    hipMemcpy(outputImage.data(), d_outputImage, size, hipMemcpyDeviceToHost);

    // stop timer and calculate bandwidth
    std::chrono::_V2::system_clock::time_point stop = std::chrono::high_resolution_clock::now();
    std::chrono::microseconds duration = std::chrono::duration_cast<std::chrono::microseconds>(start - stop);
    double bandwidth = size / duration.count() / 1e6;
    std::cout << "Global Memory Bandwidth: " << bandwidth << " bytes/second" << std::endl;

    // clean up
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

/**
 * @brief convert regular image to rotated image using the gpu shared memory
 * 
 * @param inputImage (unsigned char *) - pointer to unmodified image data
 * @param outputImage (unsigned char *) - pointer to modified image data
 * @param width (int) - width of image
 * @param height (int) - height of image
 * 
 * @return void
*/
// Define tile width as compile time const for cuda compiler
# define TILE_WIDTH 16
__global__
void sharedTransposeImage(unsigned char * input, unsigned char * output, int width, int height) {
    // Define shared memory
    __shared__ unsigned char tile[TILE_WIDTH][TILE_WIDTH + 1]; // +1 to avoid shared memory bank conflict

    // Calculate global row and column indices
    int xIndex = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int yIndex = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int index = yIndex * width + xIndex;

    // Load data into shared memory if within matrix bounds
    if (xIndex < width && yIndex < height) {
        tile[threadIdx.y][threadIdx.x] = input[index];
    }

    // Synchronize threads to ensure all data is loaded into shared memory
    __syncthreads();

    // Transpose block offset
    xIndex = blockIdx.y * TILE_WIDTH + threadIdx.x;
    yIndex = blockIdx.x * TILE_WIDTH + threadIdx.y;
    index = yIndex * height + xIndex; // Transposed index

    // Write transposed data to the output if within matrix bounds
    if (xIndex < height && yIndex < width) {
        output[index] = tile[threadIdx.x][threadIdx.y];
    }
}


/**
 * @brief convert regular image to rotated image using the gpu shared memory
 * 
 * @param inputImage (std::vector<unsigned char>) unprocessed image
 * @param outputImage (std::vector<unsigned char>) allocated vector for converted image
 * @param blocksizeX (int)
 * @param blocksizeY (int)
 * @param size (int)
 * @param width (int)
 * @param height (int)
 *
 * @return void
*/
void sharedCudaBlock(std::vector<unsigned char>& inputImage, std::vector<unsigned char>& outputImage, int blocksizeX, int blocksizeY, int width, int height, int size) {
    // cuda init/mem allocation/mem sharing
    unsigned char *d_inputImage, *d_outputImage;
    hipMalloc((void **)&d_inputImage, size);
    hipMalloc((void **)&d_outputImage, size);
    hipMemcpy(d_inputImage, inputImage.data(), size, hipMemcpyHostToDevice);

    // define block and grid sizes
    dim3 blockSize(blocksizeX, blocksizeY, 1);
    dim3 gridSize(ceil(width / blockSize.x), ceil(height / blockSize.y), 1);

    // start timer for bandwidth
    std::chrono::_V2::system_clock::time_point start = std::chrono::high_resolution_clock::now();

    // kernel function to perform a matrix transposition of the input matrix using tiling/GPU shared memory
    sharedTransposeImage<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, width, height);

    // stop timer and calculate bandwidth
    std::chrono::_V2::system_clock::time_point stop = std::chrono::high_resolution_clock::now();
    std::chrono::microseconds duration = std::chrono::duration_cast<std::chrono::microseconds>(start - stop);
    double bandwidth = size / duration.count() / 1e6;
    std::cout << "Shared Memory Bandwidth: " << bandwidth << " bytes/second" << std::endl;

    // Copy over data
    hipMemcpy(outputImage.data(), d_outputImage, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}


/**
* @brief 
* 
* @note the following values are hardcoded
*   - input file name = gc_conv_1024x1024.raw
*   - output file name = gc.raw
*   - image width = 1024
*   - image height = 1024
*   - number of channels = 3
* 
* @return error_code (int) 
* @retval 0 Success
* @retval 1 Invalid number of arguments
*/
int main(int argc, char *argv[]) {
    // sometimes hardcoding variables is okay
    std::string INPUT_FILE = "gc_1024x1024.raw";
    std::string OUTPUT_FILE = "gc.raw";
    const int WIDTH = 1024;
    const int HEIGHT = 1024;
    const int CHANNELS = 3;
    const int NUM_PIXELS = WIDTH * HEIGHT;
    const int SIZE = NUM_PIXELS * CHANNELS;

    // Check to see if valid # of params provided
    if (argc != 1) {
        std::cout << "ERROR: Inappropriate number of arguments provided!" << std::endl;
        return 1;
    }

    // init variables for start and final image
    std::vector<unsigned char> h_untouchedImage(SIZE);
    std::vector<unsigned char> h_serialConvImage(SIZE);
    std::vector<unsigned char> h_globalGPUMemoryConvImage(SIZE);
    std::vector<unsigned char> h_sharedGPUMemoryConvImage(SIZE);

    // get image data
    FILE *fp;
    fp = fopen(INPUT_FILE.c_str(), "rb");
    // make sure file exists
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << INPUT_FILE << std::endl;
        return 1;
    }
    // Read in the image file
    fread(&h_untouchedImage[0], sizeof(unsigned char), SIZE, fp);
    fclose(fp);

// ----- SERIAL IMPLEMENTATION -----
    serialTranposeImage(h_untouchedImage, h_serialConvImage, NUM_PIXELS, WIDTH, HEIGHT, CHANNELS);
    
// ----- GLOBAL MEMORY -----
    // kernel function to perform a matrix transposition of the input matrix using GPU global memory
    globalCudaBlock(h_untouchedImage, h_globalGPUMemoryConvImage, 32, 32, WIDTH, HEIGHT, SIZE * sizeof(unsigned char));

// ----- SHARED MEMORY -----
    // kernel function to perform a matrix transposition of the input matrix using tiling/GPU shared memory
    sharedCudaBlock(h_untouchedImage, h_sharedGPUMemoryConvImage, 32, 32, WIDTH, HEIGHT, SIZE * sizeof(unsigned char));

// ----- VALIDATE IMAGE PROCESSING -----
    if (!validateImageProcessing(h_serialConvImage, h_globalGPUMemoryConvImage, SIZE)) {
        std::cout << "ERROR: Global memory image processing failed!" << std::endl;
    }

    if (!validateImageProcessing(h_serialConvImage, h_sharedGPUMemoryConvImage, SIZE)) {
        std::cout << "ERROR: Global memory image processing failed!" << std::endl;
    }

// ----- SAVE IMAGES -----
    fp = fopen(OUTPUT_FILE.c_str(), "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << OUTPUT_FILE << std::endl;
        return 1;
    }
    fwrite(&h_globalGPUMemoryConvImage[0], sizeof(unsigned char), SIZE, fp);
    fclose(fp);

    
    fp = fopen("gc_global.raw", "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << "gc_global.raw" << std::endl;
        return 1;
    }
    fwrite(&h_globalGPUMemoryConvImage[0], sizeof(unsigned char), SIZE, fp);
    fclose(fp);

    fp = fopen("gc_shared.raw", "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << "gc_shared.raw" << std::endl;
        return 1;
    }
    fwrite(&h_sharedGPUMemoryConvImage[0], sizeof(unsigned char), SIZE, fp);
    fclose(fp);
}
