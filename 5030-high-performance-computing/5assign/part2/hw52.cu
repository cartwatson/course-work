/**
 * @file hw52.cpp
 * @author Carter Watson
 * @date 2023-12-01
 * @brief 
 * @todo
 * [X] read the image file
 * [X] write a CPU based serial transpose
 * [ ] write a kernel function to perform a matrix transposition of the input matrix using the GPU global memory
 * [ ] write a kernel function to perform a matrix transposition of the input matrix making use of tiling (using the GPU shared memory)
 * [X] write a function to validate the result of your transposition
 *       - (i.e., on the host CPU, comparing to a CPU based serial transpose) 
 * [ ] write code to compute the bandwidth performance of both kernels (1 and 2), report and comment the result you obtained
 * [X] save the transposed matrix to a binary file
 * 
 * @note This program is only meant to be run on University of Utah CHPC machines
 *   Helper info to compile/run on CHPC
 *     0. ssh to a CHPC machine
 *     1. salloc -n 1 -N 1 -t 0:15:00 -p notchpeak-shared-short -A notchpeak-shared-short --gres=gpu
 *     2. module load nvhpc
 *     3. nvcc -o hw52 hw52.cu
 *     4. srun ./hw52
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iomanip>
// #include <cuda_runtime.h>


/**
 * @brief validate image processing
 * 
 * @param validImage (std::vector<unsigned char>&) - vector of valid image data
 * @param checkImage (std::vector<unsigned char>&) - vector of image data to check
 * @param size (int) - size of image
 * 
 * @return bool - true if image processing is valid, otherwise false
 */
bool validateImageProcessing(std::vector<unsigned char>& validImage, std::vector<unsigned char>& checkImage, int size) {
    bool valid = true;
    for (int i = 0; i < size; i++) {
        if (validImage[i] != checkImage[i]) {
            std::cout << "ERROR: Image processing failed at index " << i << std::endl;
            std::cout << "Expected: " << std::setw(3) << (int)validImage[i] << " Actual: " << std::setw(3) << (int)checkImage[i] << std::endl;
            valid = false;
        }
    }
    return valid;
}

/**
 * @brief convert regular image to rotated image
 * 
 * @param h_untouchedImage (std::vector<unsigned char>&) - vector of unmodified image data
 * @param h_serialConvImage (std::vector<unsigned char>&) - vector of serially modified image data
 * @param numPixels (int) - number of pixels in image
 * @param width (int) - width of image
 * @param height (int) - height of image
 * @param channels (int) - number of channels in image
 * 
 * @return void
 */
void serial_tranpose(std::vector<unsigned char>& h_untouchedImage, std::vector<unsigned char>& h_serialConvImage, int numPixels, int width, int height, int channels) {
    // this is whats happening at a basic level
    // for x = 0 to N:
    //    for y = 0 to M:
    //      a_t[x][y] = a[y][x]
    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) { 
            for (int c = 0; c < channels; c++) {
                h_serialConvImage[x * height * channels + y * channels + c] = h_untouchedImage[y * width * channels + x * channels + c];
            }
        }
    }
}


/**
* @brief 
* 
* @note the following values are hardcoded
*   - input file name = gc_conv_1024x1024.raw
*   - output file name = gc.raw
*   - image width = 1024
*   - image height = 1024
*   - number of channels = 3
* 
* @return error_code (int) 
* @retval 0 Success
* @retval 1 Invalid number of arguments
*/
int main(int argc, char *argv[]) {
    // sometimes hardcoding variables is okay
    std::string INPUT_FILE = "gc_1024x1024.raw";
    std::string OUTPUT_FILE = "gc.raw";
    const int WIDTH = 1024;
    const int HEIGHT = 1024;
    const int CHANNELS = 3;
    const int NUM_PIXELS = WIDTH * HEIGHT;

    // Check to see if valid # of params provided
    if (argc != 1) {
        std::cout << "ERROR: Inappropriate number of arguments provided!" << std::endl;
        return 1;
    }

    // init variables for start and final image
    std::vector<unsigned char> h_untouchedImage(NUM_PIXELS * CHANNELS);
    std::vector<unsigned char> h_serialConvImage(NUM_PIXELS * CHANNELS);
    std::vector<unsigned char> h_convImage(NUM_PIXELS * CHANNELS);

    // get image data
    FILE *fp;
    fp = fopen(INPUT_FILE.c_str(), "rb");
    // make sure file exists
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << INPUT_FILE << std::endl;
        return 1;
    }
    // Read in the image file
    fread(&h_untouchedImage[0], sizeof(unsigned char), NUM_PIXELS * CHANNELS, fp);
    fclose(fp);

    serial_tranpose(h_untouchedImage, h_serialConvImage, NUM_PIXELS, WIDTH, HEIGHT, CHANNELS);
    
    // 

    

    // validate image processing
    validateImageProcessing(h_serialConvImage, h_convImage, NUM_PIXELS * CHANNELS);
    validateImageProcessing(h_serialConvImage, h_convImage, NUM_PIXELS * CHANNELS);

    // Save the converted image in a binary file named gc.raw
    fp = fopen(OUTPUT_FILE.c_str(), "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << OUTPUT_FILE << std::endl;
        return 1;
    }
    fwrite(&h_convImage[0], sizeof(unsigned char), NUM_PIXELS * CHANNELS, fp);
    fclose(fp);
}
