#include "hip/hip_runtime.h"
/**
 * @file hw51.cpp
 * @author Carter Watson
 * @date nov-20-2023
 * @brief CUDA program to convert a color image to gray scale
 * @todo
 * PART 1: CUDA program to convert a color image to gray scale
 * - [X] read the image file
 * - [ ] write a kernal function to perform the gray scale conversion using CUDA
 *   - remember to treat the matrices as 1D arrays and use the corresponding linearization of indexes for every memory access you perform
 *   - provide only the program with your comments as your solution. No need to attach input or output files
 * - [X] save the converted image in a binary file named gc.raw
 * 
 * PART 2: GPU Information
 * - Pick a particular GPU (e.g., from the ones available on a CHPC cluster) and report the information
 *   - [ ] Max num of threads per SM
 *   - [ ] Assume max num of blocks per SM is 8
 * - [ ] choose three different block sizes and explain analytically how the different block sizes should affect the performance of the application made in Part 1
 * - [ ] report experimental results using the three different block sizes
 *   - (i.e., run the kernel 200 times to assess the time accurately)
 * 
 * @note This program is only meant to be run on University of Utah CHPC machines
 */
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iomanip>

#include "device_query.cu"


/**
 * @brief CUDA function to convert a color image to gray scale
 * 
 * @param rgbImage (std::vector<unsigned char>) unprocessed image
 * @param width (int) width of image
 * @param height (int) height of image
 * 
 * @return processed_image (std::vector<unsigned char>) processed image
 * 
 * @note This function is primarily from lecture notes
 */
std::vector<unsigned char> RGBToGrayscale(std::vector<unsigned char> rgbImage, int width, int height, int channels)
{
    std::vector<unsigned char> grayImage(width*height);
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;

    if (Col < width && Row < height) {
        // get 1D coordinate for the grayscale image
        int grayOffset = Row*width + Col;

        // one can think of the RGB image having CHANNEL times columns of the gray scale image
        int rgbOffset = grayOffset * channels;
        unsigned char r = rgbImage[rgbOffset ]; // red value for pixel
        unsigned char g = rgbImage[rgbOffset + 1]; // green value for pixel
        unsigned char b = rgbImage[rgbOffset + 2]; // blue value for pixel

        // perform the rescaling and store it
        // NOTE: this implementation may result in rounding errors
        grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }

    return grayImage;
}


/**
* @brief 
* 
* @param input_file (str) name of the input file
* @param output_file (str) name of the output file
* @note param ordering: <input_file> <output_file>
* @note both params are optional, if not provided, default values will be used
* @note image is assumed to be 1024x1024 and have three channels, this is hardcoded
* 
* @return error_code (int) 
* @retval 0 Success
* @retval 1 Invalid number of arguments
*/
int main(int argc, char *argv[]) {
    // init variables
    std::string INPUT_FILE = "gc_conv_1024x1024.raw";
    std::string OUTPUT_FILE = "gc.raw";
    const int WIDTH = 1024;
    const int HEIGHT = 1024;
    const int NUM_PIXELS = WIDTH * HEIGHT;
    const int CHANNELS = 3;

    // Check to see if valid # of params provided
    if (argc > 4) {
        std::cout << "ERROR: Inappropriate number of arguments provided!" << std::endl;
        return 1;
    }
    // Assign file names as provided
    if (argc == 3) { OUTPUT_FILE = argv[2]; }
    if (argc >= 2) { INPUT_FILE = argv[1]; }
    
    // PART 1
    // init variables for reading in image
    std::vector<unsigned char> unprocessed_image(NUM_PIXELS * CHANNELS); // multiplied by 3 for RGB channels
    FILE *fp;
    fp = fopen(INPUT_FILE.c_str(), "rb");
    // make sure file exists
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << INPUT_FILE << std::endl;
        return 1;
    }
    // Read in the image file
    fread(&unprocessed_image[0], sizeof(unsigned char), NUM_PIXELS, fp);
    fclose(fp);

    // Convert the image to grayscale
    std::vector<unsigned char> processed_image = RGBToGrayscale(unprocessed_image, WIDTH, HEIGHT, CHANNELS);

    // Save the converted image in a binary file named gc.raw
    fp = fopen(OUTPUT_FILE.c_str(), "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << OUTPUT_FILE << std::endl;
        return 1;
    }
    fwrite(&processed_image[0], sizeof(unsigned char), NUM_PIXELS, fp);
    fclose(fp);


    // PART 2
    // Pick a particular GPU (e.g., from the ones available on a CHPC cluster) and report the information
        // Max num of threads per SM
        // Assume max num of blocks per SM is 8
    // use device query.cu to get info

    // Choose three different block sizes and explain analytically how the different block sizes should affect the performance of the application made in Part 1
    // Report experimental results using the three different block sizes


    
    return 0;
}
