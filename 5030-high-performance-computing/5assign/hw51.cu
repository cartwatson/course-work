/**
 * @file hw51.cpp
 * @author Carter Watson
 * @date nov-20-2023
 * @brief CUDA program to convert a color image to gray scale
 * @todo
 * PART 1: CUDA program to convert a color image to gray scale
 * - [X] read the image file
 * - [ ] write a kernal function to perform the gray scale conversion using CUDA
 *   - remember to treat the matrices as 1D arrays and use the corresponding linearization of indexes for every memory access you perform
 *   - provide only the program with your comments as your solution. No need to attach input or output files
 * - [X] save the converted image in a binary file named gc.raw
 * 
 * PART 2: GPU Information
 * - Pick a particular GPU (e.g., from the ones available on a CHPC cluster) and report the information
 *   - [ ] Max num of threads per SM
 *   - [ ] Assume max num of blocks per SM is 8
 * - [ ] choose three different block sizes and explain analytically how the different block sizes should affect the performance of the application made in Part 1
 * - [ ] report experimental results using the three different block sizes
 *   - (i.e., run the kernel 200 times to assess the time accurately)
 * 
 * @note This program is only meant to be run on University of Utah CHPC machines
 */
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h>


/**
 * @brief CUDA function to convert a color image to gray scale
 * 
 * @param grayImage (unsigned char *) processed image
 * @param rgbImage (unsigned char *) unprocessed image
 * @param width (int) width of image
 * @param height (int) height of image
 * @param CHANNELS (int) number of channels per pixel
 * 
 * @note This function is directly from lecture notes
 */
__global__
void RGBToGrayscale(unsigned char * grayImage, unsigned char * rgbImage, int width, int height, int CHANNELS)
{
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;

    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);//DEBUG
    if (Col < width && Row < height) {
        printf("inside loop\n");//DEBUG
        // get 1D coordinate for the grayscale image
        int grayOffset = Row*width + Col;

        // one can think of the RGB image having
        // CHANNEL times columns of the gray scale image
        int rgbOffset = grayOffset*CHANNELS;
        unsigned char r = rgbImage[rgbOffset ]; // red value for pixel
        unsigned char g = rgbImage[rgbOffset + 1]; // green value for pixel
        unsigned char b = rgbImage[rgbOffset + 2]; // blue value for pixel
        
        // perform the rescaling and store it
        // We multiply by floating point constants
        grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}


/**
 * @brief 
 * 
 * @param h_rgbImage (std::vector<unsigned char>)
 * @param h_grayImage (std::vector<unsigned char>)
 * @param blocksizeX (int)
 * @param blocksizeY (int)
 * @param NUM_PIXELS (int)
 * @param WIDTH (int)
 * @param HEIGHT (int)
 * @param CHANNELS (int)
 *
*/
void cudaBlock(std::vector<unsigned char> h_rgbImage, std::vector<unsigned char> h_grayImage, int blocksizeX, int blocksizeY, int NUM_PIXELS, int WIDTH, int HEIGHT, int CHANNELS) {
    // cuda init/mem allocation/mem sharing
    unsigned char *d_rgbImage, *d_grayImage;
    hipMalloc((void **)&d_rgbImage, NUM_PIXELS * CHANNELS * sizeof(unsigned char));
    hipMalloc((void **)&d_grayImage, NUM_PIXELS * sizeof(unsigned char));
    hipMemcpy(d_rgbImage, h_rgbImage.data(), NUM_PIXELS * CHANNELS * sizeof(unsigned char), hipMemcpyHostToDevice);

    // define block and grid sizes
    dim3 blockSize(blocksizeX, blocksizeY);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);

    printf("launching kernel\n");//DEBUG
    // Convert the image to grayscale
    RGBToGrayscale<<<gridSize, blockSize>>>(d_grayImage, d_rgbImage, WIDTH, HEIGHT, CHANNELS);

    // output error if there is an issue with the kernal function
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // sync up cuda and data
    hipDeviceSynchronize();
    printf("sync'd kernel\n");//DEBUG
    hipMemcpy(h_grayImage.data(), d_grayImage, NUM_PIXELS * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // clean up
    hipFree(d_rgbImage);
    hipFree(d_grayImage);
}


/**
* @brief 
* 
* @note the following values are hardcoded
*   - input file name = gc_conv_1024x1024.raw
*   - output file name = gc.raw
*   - image width = 1024
*   - image height = 1024
*   - number of channels = 3
* 
* @return error_code (int) 
* @retval 0 Success
* @retval 1 Invalid number of arguments
*/
int main(int argc, char *argv[]) {
    // sometimes hardcoding variables is okay
    std::string INPUT_FILE = "gc_conv_1024x1024.raw";
    std::string debug_file = "gc_temp.raw";
    std::string OUTPUT_FILE = "gc.raw";
    const int WIDTH = 1024;
    const int HEIGHT = 1024;
    const int CHANNELS = 3;
    const int NUM_PIXELS = WIDTH * HEIGHT;

    // Check to see if valid # of params provided
    if (argc != 1) {
        std::cout << "ERROR: Inappropriate number of arguments provided!" << std::endl;
        return 1;
    }
    
    // PART 1
    // init image variables
    std::vector<unsigned char> h_rgbImage(NUM_PIXELS * CHANNELS);
    std::vector<unsigned char> h_grayImage(NUM_PIXELS);

    // get image data
    FILE *fp;
    fp = fopen(INPUT_FILE.c_str(), "rb");
    // make sure file exists
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << INPUT_FILE << std::endl;
        return 1;
    }
    // Read in the image file
    fread(&h_rgbImage[0], sizeof(unsigned char), NUM_PIXELS, fp);
    fclose(fp);

    // run cuda kernal
    cudaBlock(h_rgbImage, h_grayImage, 16, 16, NUM_PIXELS, WIDTH, HEIGHT, CHANNELS);

    // Save the converted image in a binary file named gc.raw
    fp = fopen(OUTPUT_FILE.c_str(), "wb");
    if (fp == NULL) {
        std::cout << "ERROR: Could not open file " << OUTPUT_FILE << std::endl;
        return 1;
    }
    fwrite(&h_grayImage[0], sizeof(unsigned char), NUM_PIXELS, fp);
    fclose(fp);


    // PART 2
    // Pick a particular GPU (e.g., from the ones available on a CHPC cluster) and report the information
        // Max num of threads per SM
        // Assume max num of blocks per SM is 8
    // use device query.cu to get info

    // Choose three different block sizes and explain analytically how the different block sizes should affect the performance of the application made in Part 1
    // Report experimental results using the three different block sizes
    
    return 0;
}
